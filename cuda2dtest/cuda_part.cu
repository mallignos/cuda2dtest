#include "hip/hip_runtime.h"
#include "cuda_part.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

#ifdef __HIPCC__
#define KERNEL_ARGS2(grid,block) <<< grid , block >>>
#else
#define KERNEL_ARGS2(grid, block)
#endif


__global__
void cuda_run(void *arr, size_t pitch) {
	int id = ((int)blockIdx.x) * ((int)blockDim.x) + (int)(threadIdx.x);
	int x = id / 5;
	int y = id % 5;
	float a, b;

	if (id < 25) {
		float *row_a = (float *)((char *)arr + y * pitch);
		a = row_a[x];
		
		row_a[x] += 1;
		b = row_a[x];
		
		printf("bye %d %f %f\n", id, a, b);
	}
	
}


void setup(float **h_arr) {

	

	void *d_arr;
	size_t pitch;
	hipError_t e;
	e = hipMallocPitch(&d_arr, &pitch, 5 * sizeof(float), 5 );
	if (e != hipSuccess) {
		std::cout << "Memalloc fail\n";
	}
	std::cout << pitch << "\n";

	e = hipMemcpy2D(d_arr, pitch, *h_arr, 5 * sizeof(float), 5 * sizeof(float) , 5 , hipMemcpyHostToDevice);
	//e = hipMemset2D(d_arr, pitch, 1, 5 * sizeof(float), 5);
	if (e != hipSuccess) {
		std::cout << "Memcpy fail\n";
	}

	cuda_run KERNEL_ARGS2(1, 64) (d_arr,pitch);

	e = hipMemcpy2D(*h_arr, 5 * sizeof(float), d_arr, pitch, 5 * sizeof(float), 5 , hipMemcpyDeviceToHost);
	if (e != hipSuccess) {
		std::cout << "2nd Memcpy fail\n";
	}
	hipFree(d_arr);

}